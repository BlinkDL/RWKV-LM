#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ _s,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;
    _s += h*_N_*_N_ + i*_N_;

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    float state[_N_];

    __syncthreads();
    u[i] = float(_u[i]);
    __syncthreads();
    for (int j = 0; j < _N_; j++) {
        state[j] = float(_s[j]);
    }

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        w[i] = __expf(-__expf(float(_w[t])));
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
}

template <typename F>
__global__ void kernel_backward_111(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ _s, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const _gu, F *__restrict__ const _gs)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;
    _s += h*_N_*_N_ + i;

    __shared__ float u_[_N_];
    __shared__ float r[_N_], k[_N_], v[_N_], w_[_N_], gy[_N_];
    __syncthreads();
    u_[i] = float(_u[i]);
    __syncthreads();

    const float u = u_[i];

    float state[_N_], scccc[_N_] = {0}, sdddd[_N_] = {0}, sssss[_N_] = {0}, swwww[_N_];
    for (int j = 0; j < _N_; j++) {
        state[j] = float(_s[j*_N_]);
        swwww[j] = 1.0;
    }

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_T_1 = t_0 + (T-1)*C;
    const int t_T = t_0 + T*C;

    float gu = 0;
    for (int t = t_0; t < t_T; t += C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float k = float(_k[t]);
        const float w = __expf(-__expf(float(_w[t])));
        float gr = 0, gu_ = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = state[j];
            float x = k * v[j];

            gr += (u * x + s) * gy[j];
            gu_ += x * gy[j];
            s = s * w + x;
        }
        _gr[t] = F(gr);
        gu += float(_r[t]) * gu_;
    }
    _gu[b*C + h*_N_ + i] = F(gu);

    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float rr = float(_r[t]);
        const float w = __expf(-__expf(float(_w[t])));
        float gk = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = rr * gy[j];
            
            gk += (u * x + s) * v[j];
            s = x + s * w;
        }
        _gk[t] = F(gk);
    }

    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        w_[i] = __expf(-__expf(float(_w[t])));
        __syncthreads();

        const float gyy = float(_gy[t]);
        float gv = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = sdddd[j];
            float x = gyy * r[j];
            
            gv += (u_[j] * x + s) * k[j];
            s = x + s * w_[j];
        }
        _gv[t] = F(gv);
    }

    for (int t = t_0; t < t_T; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        w_[i] = __expf(-__expf(float(_w[t])));
        __syncthreads();

        const float gyy = float(_gy[t]);

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& w = swwww[j];
            sssss[j] += gyy * w * r[j];
            w *= w_[j];
        }
    }
    for (int j = 0; j < _N_; j++)
        _gs[b*H*_N_*_N_ + h*_N_*_N_ + i*_N_ + j] = F(sssss[j]);
}

template <typename F>
__global__ void kernel_backward_222(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ _s, const F *__restrict__ const _gy,
    F *__restrict__ const _gw)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _s += h*_N_*_N_ + i;

    __shared__ float v[_N_], gy[_N_];
    float state[_N_], saaaa[_N_] = {0}, sbbbb[_T_-1] = {0}, scccc[_N_] = {0};
    for (int j = 0; j < _N_; j++) {
        state[j] = float(_s[j*_N_]);
    }

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_1 = t_0 + C;
    const int t_2 = t_0 + 2*C;
    const int t_T_1 = t_0 + (T-1)*C;

    for (int t = t_T_1; t > t_1; t -= C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t-2*C]);
        __syncthreads();

        const float r = float(_r[t]);
        const float w = __expf(-__expf(float(_w[t-C])));
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = saaaa[j];
            s = (s + r * gy[j]) * w;
            sum += s * v[j];
        }
        sbbbb[(t-t_1)/C] = sum * float(_k[t-2*C]);
    }
    {
        __syncthreads();
        gy[i] = float(_gy[t_1]);
        __syncthreads();

        const float r = float(_r[t_1]);
        const float w = __expf(-__expf(float(_w[t_0])));
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = saaaa[j];
            s = (s + r * gy[j]) * w;
            sum += s * state[j];
        }
        sbbbb[0] = sum;
    }

    float sss = sbbbb[0];
    _gw[t_0] = F(sss * -__expf(float(_w[t_0])));

    {
        __syncthreads();
        gy[i] = float(_gy[t_1]);
        __syncthreads();

        const float w = __expf(-__expf(float(_w[t_0])));
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            s = (s + state[j]) * w;
            sum += s * gy[j];
        }
        sss += sbbbb[1] - (sum * float(_r[t_1]));
        _gw[t_1] = F(sss * -__expf(float(_w[t_1])));
    }
    for (int t = t_2; t < t_T_1; t += C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t-2*C]);
        __syncthreads();

        const float w = __expf(-__expf(float(_w[t-C])));
        const float k = float(_k[t-2*C]);
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            s = (s + k * v[j]) * w;
            sum += s * gy[j];
        }
        sss += sbbbb[(t-t_0)/C] - (sum * float(_r[t]));
        _gw[t] = F(sss * -__expf(float(_w[t])));
    }
    _gw[t_T_1] = 0;
}

void cuda_forward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, bf16 *w, bf16 *u, bf16 *z, bf16 *y)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, z, y);
}

void cuda_backward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, bf16 *w, bf16 *u, bf16 *z, bf16 *gy, bf16 *gr, bf16 *gk, bf16 *gv, bf16 *gw, bf16 *gu, bf16 *gs)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward_111<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, z, gy, gr, gk, gv, gu, gs);
    kernel_backward_222<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, z, gy, gw);
}
