#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#define MIN_VALUE (-1e38)
typedef at::BFloat16 bf16;

__global__ void kernel_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const bf16 *__restrict__ const _u, const bf16 *__restrict__ const _k, const bf16 *__restrict__ const _v,
                               bf16 *__restrict__ const _y) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    float u = float(_u[_c]);
    float w = _w[_c];
    const bf16 *__restrict__ const k = _k + _offset;
    const bf16 *__restrict__ const v = _v + _offset;
    bf16 *__restrict__ const y = _y + _offset;

    // aa and bb are running sums divided by exp(pp) (to avoid overflow)
    float aa = 0, bb = 0, pp = MIN_VALUE;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);

        float ww = u + kk;
        float p = max(pp, ww);
        float e1 = exp(pp - p);
        float e2 = exp(ww - p);
        y[ii] = bf16((e1 * aa + e2 * vv) / (e1 * bb + e2));
        
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
}

__global__ void kernel_backward(const int B, const int T, const int C,
                                const float *__restrict__ const _w, const bf16 *__restrict__ const _u, const bf16 *__restrict__ const _k, const bf16 *__restrict__ const _v,
                                const bf16 *__restrict__ const _y, const bf16 *__restrict__ const _gy,
                                bf16 *__restrict__ const _gw, bf16 *__restrict__ const _gu, bf16 *__restrict__ const _gk, bf16 *__restrict__ const _gv) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    float u = float(_u[_c]);
    float w = _w[_c];
    const bf16 *__restrict__ const k = _k + _offset;
    const bf16 *__restrict__ const v = _v + _offset;
    const bf16 *__restrict__ const y = _y + _offset;
    const bf16 *__restrict__ const gy = _gy + _offset;
    bf16 *__restrict__ const gk = _gk + _offset;
    bf16 *__restrict__ const gv = _gv + _offset;

    float q[Tmax], r[Tmax];

    float gw = 0, gu = 0, aa = 0, bb = 0, ga = 0, gb = 0, pp = MIN_VALUE;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        const float yy = float(y[ii]);

        float ww = u + kk;
        float p = max(pp, ww);
        float e1 = exp(pp - p);
        float e2 = exp(ww - p);
        const float qq = float(gy[ii]) / (e1 * bb + e2);
        gw += (ga - gb * yy) * e1 * qq;
        gu += (vv - yy) * e2 * qq;
        q[i] = qq;
        r[i] = ww - p;

        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        ga = e1 * (aa + ga);
        gb = e1 * (bb + gb);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] = bf16(gw * _w[_c]); // multiply by w because of w -> -exp(w) in python forward()
    _gu[_offsetBC] = bf16(gu);

    aa = 0, bb = 0, pp = MIN_VALUE;
    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        const float yy = float(y[ii]);
        const float qq = q[i];
        const float rr = r[i];

        float e1 = qq * exp(rr);
        float e2 = exp(kk + pp);
        gk[ii] = bf16(e1 * (vv - yy) + e2 * (aa * vv + bb));
        gv[ii] = bf16(e1 + e2 * aa);

        const float ww = w + pp;
        const float www = rr - u - kk;
        const float p = max(ww, www);
        e1 = exp(ww - p);
        e2 = qq * exp(www - p);
        aa = e1 * aa + e2;
        bb = e1 * bb - e2 * yy;
        pp = p;
    }
}

void cuda_forward(int B, int T, int C, float *w, bf16 *u, bf16 *k, bf16 *v, bf16 *y) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y);
}

void cuda_backward(int B, int T, int C, float *w, bf16 *u, bf16 *k, bf16 *v, bf16 *y, bf16 *gy, bf16 *gw, bf16 *gu, bf16 *gk, bf16 *gv) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, gy, gw, gu, gk, gv);
}
