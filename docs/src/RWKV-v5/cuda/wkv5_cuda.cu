#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    float state[_N_] = {0};

    __syncthreads();
    w[i] = _w[i];
    u[i] = float(_u[i]);
    __syncthreads();

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
}

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const float *__restrict__ __w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const _gw, F *__restrict__ const _gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;
    __w += h*_N_;

    __shared__ float w_[_N_], u_[_N_];
    __shared__ float r[_N_], k[_N_], v[_N_], gy[_N_];
    __syncthreads();
    w_[i] = _w[i];
    u_[i] = float(_u[i]);
    __syncthreads();

    const float w = w_[i];
    const float ww = __w[i];
    const float u = u_[i];

    float state[_N_] = {0}, saaaa[_N_] = {0}, sbbbb[_N_] = {0}, scccc[_N_] = {0}, sdddd[_N_] = {0};

    float gw = 0, gu = 0;
    const int t000 = b*T*C + h*_N_ + i;
    const int t111 = (b+1)*T*C + h*_N_ + i;
    const int t222 = t111 - 2*C;

    for (int t = t000; t < t111; t += C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float k = float(_k[t]);
        float gr = 0, gu_ = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = state[j];
            float x = k * v[j];

            gr += (u * x + s) * gy[j];
            gu_ += x * gy[j];
            s = s * w + x;
        }
        _gr[t] = F(gr);
        gu += float(_r[t]) * gu_;
    }
    _gu[b*C + h*_N_ + i] = F(gu);
    
    for (int t = t000; t < t222; t += C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t + 2*C]);
        __syncthreads();

        const float k = float(_k[t]);
        float gw_ = 0;
        
        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = saaaa[j];
            float& s2 = sbbbb[j];
            float x = k * v[j];
            
            float tmp = w * (x + s);
            s = tmp;
            s2 = tmp + w * s2;
            gw_ += s2 * gy[j];
        }
        gw += float(_r[t + 2*C]) * gw_;
    }    
    _gw[b*C + h*_N_ + i] = F(ww * gw);

    for (int t = t111 - C; t >= t000; t -= C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float rr = float(_r[t]);
        float gk = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = rr * gy[j];
            
            gk += (u * x + s) * v[j];
            s = x + s * w;
        }
        _gk[t] = F(gk);
    }

    for (int t = t111 - C; t >= t000; t -= C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float gyy = float(_gy[t]);
        float gv = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = sdddd[j];
            float x = gyy * r[j];
            
            gv += (u_[j] * x + s) * k[j];
            s = x + s * w_[j];
        }
        _gv[t] = F(gv);
    }
}

void cuda_forward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, bf16 *u, bf16 *y)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, float *ww, bf16 *u, bf16 *gy, bf16 *gr, bf16 *gk, bf16 *gv, bf16 *gw, bf16 *gu)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}
