#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define MIN_VALUE (-1e38)

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C,
                               const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v,
                               F *__restrict__ const _y) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    F *__restrict__ const y = _y + _offset;

    F p = 0, q = 0, o = MIN_VALUE;
    // p and q are running sums divided by exp(o) (to avoid overflows)
    for (int i = 0; i < T; i++) {
        const int ii = i * C;

        F no = max(o, u + k[ii]);
        F A = exp(o - no);
        F B = exp(u + k[ii] - no);
        y[ii] = (A * p + B * v[ii]) / (A * q + B);

        no = max(w + o, k[ii]);
        A = exp(w + o - no);
        B = exp(k[ii] - no);
        p = A * p + B * v[ii];
        q = A * q + B;
        o = no;
    }
}

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C,
                                const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ const _gy,
                                F *__restrict__ const _gw, F *__restrict__ const _gu, F *__restrict__ const _gk, F *__restrict__ const _gv) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    const F *__restrict__ const gy = _gy + _offset;

    F *__restrict__ const gk = _gk + _offset;
    F *__restrict__ const gv = _gv + _offset;

    F y[Tmax], z[Tmax], zexp[Tmax];

    F gw = 0, gu = 0;
    F p = 0, q = 0;
    F dpdw = 0, dqdw = 0;
    F o = MIN_VALUE;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        F no = max(o, k[ii] + u);
        F A = exp(o - no);
        F B = exp(k[ii] + u - no);

        F num = A * p + B * v[ii];
        F iden = 1 / (A * q + B);

        y[i] = num * iden;
        z[i] = iden;
        zexp[i] = k[ii] + u - no;

        gw += gy[ii] * (dpdw - dqdw * y[i]) * iden * A;
        gu += gy[ii] * (v[ii] - y[i]) * B * iden;

        no = max(w + o, k[ii]);
        A = exp(w + o - no);
        B = exp(k[ii] - no);
        dpdw = A * (p + dpdw);
        dqdw = A * (q + dqdw);
        p = A * p + B * v[ii];
        q = A * q + B;
        o = no;
    }

    F gp = 0, gq = 0;
    o = MIN_VALUE;
    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        F A = gy[ii] * z[i] * exp(zexp[i]);
        F B = exp(k[ii] + o);
        gk[ii] = A * (v[ii] - y[i]) + B * (gp * v[ii] + gq);
        gv[ii] = A + B * gp;

        F no = max(w + o, zexp[i] - k[ii] - u);
        A = exp(w + o - no);
        B = gy[ii] * z[i] * exp(zexp[i] - k[ii] - u - no);
        gp = A * gp + B;
        gq = A * gq - B * y[i];
        o = no;
    }

    // Multiply by w because the w -> -exp(w) preprocessing is halfway in the backwards pass, even though it's not in the forward pass
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] += gw * _w[_c];
    _gu[_offsetBC] += gu;
}

void cuda_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y);
}

void cuda_backward(int B, int T, int C, float *w, float *u, float *k, float *v, float *gy, float *gw, float *gu, float *gk, float *gv) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, gy, gw, gu, gk, gv);
}
