#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    float state[_N_] = {0};

    __syncthreads();
    u[i] = float(_u[i]);
    __syncthreads();

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        w[i] = __expf(-__expf(float(_w[t])));
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
}

template <typename F>
__global__ void kernel_backward_101(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;

    __shared__ float v[_N_], gy[_N_];

    const float u = float(_u[h*_N_ + i]);

    float state[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_T = t_0 + T*C;

    float gu = 0;
    for (int t = t_0; t < t_T; t += C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float k = float(_k[t]);
        const float w = __expf(-__expf(float(_w[t])));
        float gr = 0, gu_ = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = state[j];
            float x = k * v[j];

            gr += (u * x + s) * gy[j];
            gu_ += x * gy[j];
            s = s * w + x;
        }
        _gr[t] = F(gr);
        gu += float(_r[t]) * gu_;
    }
    _gu[b*C + h*_N_ + i] = F(gu);
}

template <typename F>
__global__ void kernel_backward_102(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gk)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;

    __shared__ float v[_N_], gy[_N_];

    const float u = float(_u[h*_N_ + i]);

    float scccc[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_T_1 = t_0 + (T-1)*C;

    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float rr = float(_r[t]);
        const float w = __expf(-__expf(float(_w[t])));
        float gk = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = rr * gy[j];
            
            gk += (u * x + s) * v[j];
            s = x + s * w;
        }
        _gk[t] = F(gk);
    }
}

template <typename F>
__global__ void kernel_backward_103(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gv)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;

    __shared__ float u_[_N_], r[_N_], k[_N_], w_[_N_];
    __syncthreads();
    u_[i] = float(_u[i]);
    __syncthreads();

    float sdddd[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_T_1 = t_0 + (T-1)*C;

    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        w_[i] = __expf(-__expf(float(_w[t])));
        __syncthreads();

        const float gyy = float(_gy[t]);
        float gv = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = sdddd[j];
            float x = gyy * r[j];
            
            gv += (u_[j] * x + s) * k[j];
            s = x + s * w_[j];
        }
        _gv[t] = F(gv);
    }
}

template <typename F>
__global__ void kernel_backward_201(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gw)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;

    __shared__ float v[_N_], gy[_N_];
    float saaaa[_N_] = {0}, sbbbb[_T_-2] = {0}, scccc[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_1 = t_0 + C;
    const int t_2 = t_0 + 2*C;
    const int t_T_1 = t_0 + (T-1)*C;

    for (int t = t_T_1; t > t_1; t -= C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t-2*C]);
        __syncthreads();

        const float r = float(_r[t]);
        const float w = __expf(-__expf(float(_w[t-C])));
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = saaaa[j];
            float x = r * gy[j];
            s = (s + x) * w;
            sum += s * v[j];
        }
        sbbbb[(t-t_2)/C] = sum * float(_k[t-2*C]);
    }

    float sss = sbbbb[0];
    _gw[t_0] = 0;
    _gw[t_1] = F(sss * -__expf(float(_w[t_1])));

    for (int t = t_2; t < t_T_1; t += C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t-2*C]);
        __syncthreads();

        const float w = __expf(-__expf(float(_w[t-C])));
        const float k = float(_k[t-2*C]);
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = k * v[j];
            s = (s + x) * w;
            sum += s * gy[j];
        }
        sss += sbbbb[(t-t_1)/C] - (sum * float(_r[t]));
        _gw[t] = F(sss * -__expf(float(_w[t])));
    }
    _gw[t_T_1] = 0;
}

void cuda_forward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, bf16 *w, bf16 *u, bf16 *y)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, bf16 *w, bf16 *u, bf16 *gy, bf16 *gr, bf16 *gk, bf16 *gv, bf16 *gw, bf16 *gu)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward_101<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gr, gu);
    kernel_backward_102<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gk);
    kernel_backward_103<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gv);
    kernel_backward_201<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gw);
}
