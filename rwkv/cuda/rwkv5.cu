#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;
typedef at::Half fp16;
typedef float fp32;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H, float *__restrict__ _state,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;
    _state += h*_N_*_N_ + i*_N_; // wrong if B > 1 !!!

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    
    float state[_N_];
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        state[j] = _state[j];
    
    __syncthreads();
    u[i] = float(_u[i]);
    w[i] = _w[i];
    __syncthreads();

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        _state[j] = state[j];
}

void cuda_forward_bf16(int B, int T, int C, int H, float *state, bf16 *r, bf16 *k, bf16 *v, float *w, bf16 *u, bf16 *y)
{
    assert(H*_N_ == C);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, u, y);
}
void cuda_forward_fp16(int B, int T, int C, int H, float *state, fp16 *r, fp16 *k, fp16 *v, float *w, fp16 *u, fp16 *y)
{
    assert(H*_N_ == C);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, u, y);
}
void cuda_forward_fp32(int B, int T, int C, int H, float *state, fp32 *r, fp32 *k, fp32 *v, float *w, fp32 *u, fp32 *y)
{
    assert(H*_N_ == C);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, k, v, w, u, y);
}
