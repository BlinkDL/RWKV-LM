#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#define MIN_VALUE (-1e38)
typedef at::Half fp16;
__half *cast(fp16 *ptr) {
    return reinterpret_cast<__half *>(ptr);
}

template <typename F>
__global__ void kernel_wkv_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const float *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v,
                               F *__restrict__ const _y, float *__restrict__ const _aa, float *__restrict__ const _bb, float *__restrict__ const _pp) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _state_offset = _b * C + _c;

    float u = _u[_c];
    float w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    F *__restrict__ const y = _y + _offset;

    float aa = _aa[_state_offset];
    float bb = _bb[_state_offset];
    float pp = _pp[_state_offset];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        float ww = u + kk;
        float p = max(pp, ww);
        float e1 = exp(pp - p);
        float e2 = exp(ww - p);
        y[ii] = F((e1 * aa + e2 * vv) / (e1 * bb + e2));
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
    _aa[_state_offset] = aa;
    _bb[_state_offset] = bb;
    _pp[_state_offset] = pp;
}

template <typename F>
void cuda_wkv_forward(int B, int T, int C, float *w, float *u, F *k, F *v, F *y, float *aa, float *bb, float *pp) {
    dim3 threadsPerBlock( min(C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_wkv_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, aa, bb, pp);
}

template void cuda_wkv_forward<fp16>(
    int B, int T, int C,
    float *w, float *u, fp16 *k, fp16 *v, fp16 *y,
    float *aa, float *bb, float *pp);
template void cuda_wkv_forward<float>(
    int B, int T, int C,
    float *w, float *u, float *k, float *v, float *y,
    float *aa, float *bb, float *pp);

__global__ void kernel_mm_seq_fp32i8(
    const int B, const int N, const int M,
    const float *__restrict__ const x, const int x_stride,
    const uint8_t *__restrict__ const w, const int w_stride,
    const float *__restrict__ const mx,
    const float *__restrict__ const rx,
    const float *__restrict__ const my,
    const float *__restrict__ const ry,
    float *__restrict__ const y, const int y_stride) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && k < M) {
        float y_local = 0;
        for (int j = 0; j < N; ++j) {
            y_local += x[i * x_stride + j] * (
                (float(w[j * w_stride + k]) + 0.5f)
                * rx[k] * ry[j] + mx[k] + my[j]
            );
        }
        y[i * y_stride + k] = y_local;
    }
}

template <typename F>
void cuda_mm8_seq(int B, int N, int M,
                  F *x, int x_stride,
                  uint8_t *w, int w_stride,
                  F *mx, F *rx,
                  F *my, F *ry,
                  F *y, int y_stride);

template <>
void cuda_mm8_seq<float>(int B, int N, int M,
                         float *x, int x_stride,
                         uint8_t *w, int w_stride,
                         float *mx, float *rx,
                         float *my, float *ry,
                         float *y, int y_stride) {
    dim3 blockSize(1, 128);
    dim3 gridSize((B + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_seq_fp32i8<<<gridSize, blockSize>>>(
        B, N, M, x, x_stride, w, w_stride,
        mx, rx, my, ry, y, y_stride);
}

__global__ void kernel_mm_seq_fp16i8(
    const int B, const int N, const int M,
    const __half *__restrict__ const x, const int x_stride,
    const uint8_t *__restrict__ const w, const int w_stride,
    const __half *__restrict__ const mx,
    const __half *__restrict__ const rx,
    const __half *__restrict__ const my,
    const __half *__restrict__ const ry,
    __half *__restrict__ const y, const int y_stride) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && k < M) {
        float y_local = 0;
        for (int j = 0; j < N; ++j) {
            y_local += __half2float(x[i * x_stride + j]) * (
                (float(w[j * w_stride + k]) + 0.5f)
                * __half2float(rx[k]) * __half2float(ry[j])
                + __half2float(mx[k]) + __half2float(my[j])
            );
        }
        y[i * y_stride + k] = __float2half(y_local);
    }
}

template <>
void cuda_mm8_seq<fp16>(int B, int N, int M,
                        fp16 *x, int x_stride,
                        uint8_t *w, int w_stride,
                        fp16 *mx, fp16 *rx,
                        fp16 *my, fp16 *ry,
                        fp16 *y, int y_stride) {
    dim3 blockSize(1, 128);
    dim3 gridSize((B + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_seq_fp16i8<<<gridSize, blockSize>>>(
        B, N, M, cast(x), x_stride, w, w_stride,
        cast(mx), cast(rx), cast(my), cast(ry), cast(y), y_stride);
}

#define MM8_ONE_JSPLIT 24
#define MM8_ONE_TILE 1024

__global__ void kernel_mm_one_fp32i8(
    const int N, const int M,
    const float *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    const float *__restrict__ const mx,
    const float *__restrict__ const rx,
    const float *__restrict__ const my,
    const float *__restrict__ const ry,
    float *__restrict__ const y) {

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (float(w[j * w_stride + k]) + 0.5f)
                * rx[k] * ry[j] + mx[k] + my[j]
            );
        }
        atomicAdd(&y[k], y_local);
    }
}

template <typename F>
void cuda_mm8_one(int N, int M,
                  F *x,
                  uint8_t *w, int w_stride,
                  F *mx, F *rx,
                  F *my, F *ry,
                  float *y);

template <>
void cuda_mm8_one<float>(int N, int M,
                        float *x,
                        uint8_t *w, int w_stride,
                        float *mx, float *rx,
                        float *my, float *ry,
                        float *y) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_one_fp32i8<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride,
        mx, rx, my, ry, y);
}

__global__ void kernel_mm_one_fp16i8(
    const int N, const int M,
    const __half *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    const __half *__restrict__ const mx,
    const __half *__restrict__ const rx,
    const __half *__restrict__ const my,
    const __half *__restrict__ const ry,
    float *__restrict__ const y) {

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += __half2float(x[j]) * (
                (float(w[j * w_stride + k]) + 0.5f)
                * __half2float(rx[k]) * __half2float(ry[j])
                + __half2float(mx[k]) + __half2float(my[j])
            );
        }
        atomicAdd(&y[k], y_local);
    }
}

template <>
void cuda_mm8_one<fp16>(int N, int M,
                        fp16 *x,
                        uint8_t *w, int w_stride,
                        fp16 *mx, fp16 *rx,
                        fp16 *my, fp16 *ry,
                        float *y) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm_one_fp16i8<<<gridSize, blockSize>>>(
        N, M, cast(x), w, w_stride,
        cast(mx), cast(rx), cast(my), cast(ry), y);
}
